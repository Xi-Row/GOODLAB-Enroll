//A kernel that achieves:
//
//
//
//
#include <hip/hip_runtime.h>  //including cuda runtime API
#include <stdio.h>

#define WARP_SIZE 32  //A warp containing 32 threads
#define EPSILON 1e-6f //a small number to avoid zero in denominator

// Warp Reduce
__device__ float warpReduceSum(float val) {
    for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2) {
        val += __shfl_down_sync(0xffffffff, val, offset); //warp reducing core
    }
    return val;
}

// LayerNorm CUDA Kernel
__global__ void layer_norm_kernel(float* __restrict__ output,      //__restrict__ make the ptr safe to
                                  const float* __restrict__ input, //improve the accessment efficiency
                                  const float* __restrict__ gamma, 
                                  const float* __restrict__ beta, 
                                  int hidden_size) {
    // get message of thread
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int lane = tid % WARP_SIZE;  // index of thread in warp
    int warp_id = tid / WARP_SIZE;  // thread belongs to which warp 

    //Calculating the number of warps
    int num_warps = blockDim.x / WARP_SIZE;
    int num_vec4 = hidden_size / 4;  // float4 vectorization

    //shared memory
    __shared__ float buffer[2][WARP_SIZE];

    //float4 vectorized access //Disposing 4 float one times
    float4* input4  = (float4*) (input + bid * hidden_size);
    float4* output4 = (float4*) (output + bid * hidden_size);
    float4* gamma4  = (float4*) gamma;
    float4* beta4   = (float4*) beta;

    float sum = 0.0f, sum_sq = 0.0f;
    float4 local_data;

    //Calculate the mean and variance
    for (int i = tid; i < num_vec4; i += blockDim.x) {
        local_data = input4[i];
        float v1 = local_data.x, v2 = local_data.y;
        float v3 = local_data.z, v4 = local_data.w;

        sum += (v1 + v2 + v3 + v4);
        sum_sq += (v1 * v1 + v2 * v2 + v3 * v3 + v4 * v4);
    }

    // Warp Reduce sum
    sum = warpReduceSum(sum);
    sum_sq = warpReduceSum(sum_sq);

    //Shared memory // double-buffered
    if (lane == 0) {
        buffer[0][warp_id] = sum;
        buffer[1][warp_id] = sum_sq;
    }
    __syncthreads();

    //only one warp dealing with the final reduction
    if (warp_id == 0) {
        sum = (tid < num_warps) ? buffer[0][tid] : 0;
        sum_sq = (tid < num_warps) ? buffer[1][tid] : 0;

        sum = warpReduceSum(sum);
        sum_sq = warpReduceSum(sum_sq);

        if (tid == 0) {
            float mean = sum / hidden_size;
            float var = sum_sq / hidden_size - mean * mean;
            buffer[0][0] = mean;
            buffer[1][0] = rsqrtf(var + EPSILON);
        }
    }
    __syncthreads();

    float mean = buffer[0][0];
    float rstd = buffer[1][0];

    //Normalized calculations
    for (int i = tid; i < num_vec4; i += blockDim.x) {
        float4 g = gamma4[i];
        float4 b = beta4[i];

        local_data.x = (local_data.x - mean) * rstd * g.x + b.x;
        local_data.y = (local_data.y - mean) * rstd * g.y + b.y;
        local_data.z = (local_data.z - mean) * rstd * g.z + b.z;
        local_data.w = (local_data.w - mean) * rstd * g.w + b.w;

        output4[i] = local_data;
    }
}

//operating Kernel
void launch_layer_norm(float* output, const float* input, 
                       const float* gamma, 
                       const float* beta, int batch, 
                       int hidden_size) {
     int threads = 256;  
    int blocks = batch;
    layer_norm_kernel<<<blocks, threads>>>(output, 
                                           input, 
                                           gamma, beta, 
                                           hidden_size);
}
